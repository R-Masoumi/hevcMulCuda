#include "hip/hip_runtime.h"
#include "gdct.cuh"

template<typename T>
__device__ inline T g_x265_clip3(T minVal, T maxVal, T a) { return g_x265_min(g_x265_max(minVal, a), maxVal); }
template<typename T>
__device__ inline T g_x265_min(T a, T b) { return a < b ? a : b; }
template<typename T>
__device__ inline T g_x265_max(T a, T b) { return a > b ? a : b; }


int16_t *src, *src2, *dst;
void cudaAlloc(int n){
	hipMalloc(&dst, n * n * sizeof(int16_t));
	hipMalloc(&src, n * n * sizeof(int16_t));
	hipMalloc(&src2, n * n * sizeof(int16_t));
}

void cudaAlloc(int n,int m){
	hipMalloc(&dst, m* n * n * sizeof(int16_t));
	hipMalloc(&src, m* n * n * sizeof(int16_t));
	hipMalloc(&src2, m* n * n * sizeof(int16_t));
}

void cudaDestroy(){
	hipFree(src);
	hipFree(src2);
	hipFree(dst);
}

__global__ void fastForwardDst(const int16_t* block, int16_t* coeff, int shift)  // input block, output coeff
{
	int c[4];
	int rnd_factor = 1 << (shift - 1);
	int i = threadIdx.x;
	// Intermediate Variables
	c[0] = block[4 * i + 0] + block[4 * i + 3];
	c[1] = block[4 * i + 1] + block[4 * i + 3];
	c[2] = block[4 * i + 0] - block[4 * i + 1];
	c[3] = 74 * block[4 * i + 2];

	coeff[i] = (int16_t)((29 * c[0] + 55 * c[1] + c[3] + rnd_factor) >> shift);
	coeff[4 + i] = (int16_t)((74 * (block[4 * i + 0] + block[4 * i + 1] - block[4 * i + 3]) + rnd_factor) >> shift);
	coeff[8 + i] = (int16_t)((29 * c[2] + 55 * c[0] - c[3] + rnd_factor) >> shift);
	coeff[12 + i] = (int16_t)((55 * c[2] - 29 * c[1] + c[3] + rnd_factor) >> shift);
}

__global__ void inversedst(const int16_t* tmp, int16_t* block, int shift)  // input tmp, output block
{
	int c[4];
	int rnd_factor = 1 << (shift - 1);
	int i = threadIdx.x;//max 4

	// Intermediate Variables
	c[0] = tmp[i] + tmp[8 + i];
	c[1] = tmp[8 + i] + tmp[12 + i];
	c[2] = tmp[i] - tmp[12 + i];
	c[3] = 74 * tmp[4 + i];

	block[4 * i + 0] = (int16_t)g_x265_clip3(-32768, 32767, (29 * c[0] + 55 * c[1] + c[3] + rnd_factor) >> shift);
	block[4 * i + 1] = (int16_t)g_x265_clip3(-32768, 32767, (55 * c[2] - 29 * c[1] + c[3] + rnd_factor) >> shift);
	block[4 * i + 2] = (int16_t)g_x265_clip3(-32768, 32767, (74 * (tmp[i] - tmp[8 + i] + tmp[12 + i]) + rnd_factor) >> shift);
	block[4 * i + 3] = (int16_t)g_x265_clip3(-32768, 32767, (55 * c[0] + 29 * c[2] - c[3] + rnd_factor) >> shift);
}

__global__ void partialButterfly16(const int16_t* src, int16_t* dst, int shift, int line)
{
	__shared__ int E[8], O[8];
	__shared__ int EE[4], EO[4];
	int EEE[2], EEO[2];
	int add = 1 << (shift - 1);
	int j = blockIdx.x;//max 16
	int k = threadIdx.x;//max 16
	src += 16 * j;
	dst += j;

	/* E and O */
	if (k < 8)
	{
		E[k] = src[k] + src[15 - k];
		O[k] = src[k] - src[15 - k];
	}

	__syncthreads();

	/* EE and EO */
	if (k < 4)
	{
		EE[k] = E[k] + E[7 - k];
		EO[k] = E[k] - E[7 - k];
	}
	__syncthreads();
	if (k == 4)//unused thread
	{
		/* EEE and EEO */
		EEE[0] = EE[0] + EE[3];
		EEO[0] = EE[0] - EE[3];
		EEE[1] = EE[1] + EE[2];
		EEO[1] = EE[1] - EE[2];

		dst[0] = (int16_t)((t16[0][0] * EEE[0] + t16[0][1] * EEE[1] + add) >> shift);
		dst[8 * line] = (int16_t)((t16[8][0] * EEE[0] + t16[8][1] * EEE[1] + add) >> shift);
		dst[4 * line] = (int16_t)((t16[4][0] * EEO[0] + t16[4][1] * EEO[1] + add) >> shift);
		dst[12 * line] = (int16_t)((t16[12][0] * EEO[0] + t16[12][1] * EEO[1] + add) >> shift);
	}
	else if (k % 4 == 2)//2 6 10 14
	{
		dst[k * line] = (int16_t)((t16[k][0] * EO[0] + t16[k][1] * EO[1] + t16[k][2] * EO[2] +
			t16[k][3] * EO[3] + add) >> shift);
	}

	else if (k % 2 == 1)
	{
		dst[k * line] = (int16_t)((t16[k][0] * O[0] + t16[k][1] * O[1] + t16[k][2] * O[2] + t16[k][3] * O[3] +
			t16[k][4] * O[4] + t16[k][5] * O[5] + t16[k][6] * O[6] + t16[k][7] * O[7] +
			add) >> shift);
	}
}

__global__ void partialButterfly32(const int16_t* src, int16_t* dst, int shift, int line)
{
	int j = blockIdx.x;// max 32
	int k = threadIdx.x;//max 32
	__shared__ int E[16], O[16];
	__shared__ int EE[8], EO[8];
	__shared__ int EEE[4], EEO[4];
	int EEEE[2], EEEO[2];
	int add = 1 << (shift - 1);

	src += 32 * j;
	dst += j;

	/* E and O*/
	if (k < 16)
	{
		E[k] = src[k] + src[31 - k];
		O[k] = src[k] - src[31 - k];
	}
	__syncthreads();

	/* EE and EO */
	if (k < 8)
	{
		EE[k] = E[k] + E[15 - k];
		EO[k] = E[k] - E[15 - k];
	}
	__syncthreads();

	/* EEE and EEO */
	if (k < 4)
	{
		EEE[k] = EE[k] + EE[7 - k];
		EEO[k] = EE[k] - EE[7 - k];
	}
	__syncthreads();



	/* EEEE and EEEO */
	EEEE[0] = EEE[0] + EEE[3];
	EEEO[0] = EEE[0] - EEE[3];
	EEEE[1] = EEE[1] + EEE[2];
	EEEO[1] = EEE[1] - EEE[2];

	if (k == 1 || k == 3){
		dst[(k - 1) * 8 * line] = (int16_t)((t32[(k - 1) * 8][0] * EEEE[0] + t32[(k - 1) * 8][1] * EEEE[1] + add) >> shift);
		dst[k * 8 * line] = (int16_t)((t32[k * 8][0] * EEEO[0] + t32[k * 8][1] * EEEO[1] + add) >> shift);
	}
	if (k % 8 == 4)//4 12 20 28
	{
		dst[k * line] = (int16_t)((t32[k][0] * EEO[0] + t32[k][1] * EEO[1] + t32[k][2] * EEO[2] +
			t32[k][3] * EEO[3] + add) >> shift);
	}
	else if (k % 4 == 2)//2 6 10 14 18 22 26 30
	{
		dst[k * line] = (int16_t)((t32[k][0] * EO[0] + t32[k][1] * EO[1] + t32[k][2] * EO[2] +
			t32[k][3] * EO[3] + t32[k][4] * EO[4] + t32[k][5] * EO[5] +
			t32[k][6] * EO[6] + t32[k][7] * EO[7] + add) >> shift);
	}
	else if (k % 2 == 1)//1 3 5 7 9 - odds 
	{
		dst[k * line] = (int16_t)((t32[k][0] * O[0] + t32[k][1] * O[1] + t32[k][2] * O[2] + t32[k][3] * O[3] +
			t32[k][4] * O[4] + t32[k][5] * O[5] + t32[k][6] * O[6] + t32[k][7] * O[7] +
			t32[k][8] * O[8] + t32[k][9] * O[9] + t32[k][10] * O[10] + t32[k][11] *
			O[11] + t32[k][12] * O[12] + t32[k][13] * O[13] + t32[k][14] * O[14] +
			t32[k][15] * O[15] + add) >> shift);
	}
}

__global__ void partialButterfly8(const int16_t* src, int16_t* dst, int shift, int line)
{
	int j = blockIdx.x;// max 8
	int k = threadIdx.x;//max 8
	__shared__ int E[4], O[4];
	int EE[2], EO[2];
	int add = 1 << (shift - 1);

	src += 8 * j;
	dst += j;

	/* E and O*/
	if (k < 4)
	{
		E[k] = src[k] + src[7 - k];
		O[k] = src[k] - src[7 - k];
	}

	__syncthreads();

	/* EE and EO */
	EE[0] = E[0] + E[3];
	EO[0] = E[0] - E[3];
	EE[1] = E[1] + E[2];
	EO[1] = E[1] - E[2];

	dst[0] = (int16_t)((t8[0][0] * EE[0] + t8[0][1] * EE[1] + add) >> shift);
	dst[4 * line] = (int16_t)((t8[4][0] * EE[0] + t8[4][1] * EE[1] + add) >> shift);
	dst[2 * line] = (int16_t)((t8[2][0] * EO[0] + t8[2][1] * EO[1] + add) >> shift);
	dst[6 * line] = (int16_t)((t8[6][0] * EO[0] + t8[6][1] * EO[1] + add) >> shift);

	dst[line] = (int16_t)((t8[1][0] * O[0] + t8[1][1] * O[1] + t8[1][2] * O[2] + t8[1][3] * O[3] + add) >> shift);
	dst[3 * line] = (int16_t)((t8[3][0] * O[0] + t8[3][1] * O[1] + t8[3][2] * O[2] + t8[3][3] * O[3] + add) >> shift);
	dst[5 * line] = (int16_t)((t8[5][0] * O[0] + t8[5][1] * O[1] + t8[5][2] * O[2] + t8[5][3] * O[3] + add) >> shift);
	dst[7 * line] = (int16_t)((t8[7][0] * O[0] + t8[7][1] * O[1] + t8[7][2] * O[2] + t8[7][3] * O[3] + add) >> shift);
}

__global__ void partialButterflyInverse4(const int16_t* src, int16_t* dst, int shift, int line)
{
	int j = threadIdx.x;
	int E[2], O[2];
	int add = 1 << (shift - 1);

	dst += 4 * j;
	src += j;

	/* Utilizing symmetry properties to the maximum to minimize the number of multiplications */
	O[0] = t4[1][0] * src[line] + t4[3][0] * src[3 * line];
	O[1] = t4[1][1] * src[line] + t4[3][1] * src[3 * line];
	E[0] = t4[0][0] * src[0] + t4[2][0] * src[2 * line];
	E[1] = t4[0][1] * src[0] + t4[2][1] * src[2 * line];

	/* Combining even and odd terms at each hierarchy levels to calculate the final spatial domain vector */
	dst[0] = (int16_t)(g_x265_clip3(-32768, 32767, (E[0] + O[0] + add) >> shift));
	dst[1] = (int16_t)(g_x265_clip3(-32768, 32767, (E[1] + O[1] + add) >> shift));
	dst[2] = (int16_t)(g_x265_clip3(-32768, 32767, (E[1] - O[1] + add) >> shift));
	dst[3] = (int16_t)(g_x265_clip3(-32768, 32767, (E[0] - O[0] + add) >> shift));
}

__global__ void partialButterflyInverse8(const int16_t* src, int16_t* dst, int shift, int line)
{
	int j = blockIdx.x;// max 8
	int k = threadIdx.x;//max 8
	__shared__ int E[4], O[4];
	int EE[2], EO[2];
	int add = 1 << (shift - 1);

	dst += 8 * j;
	src += j;

	/* Utilizing symmetry properties to the maximum to minimize the number of multiplications */
	if (k < 4)
	{
		O[k] = t8[1][k] * src[line] + t8[3][k] * src[3 * line] + t8[5][k] * src[5 * line] + t8[7][k] * src[7 * line];
	}

	if (k == 5)//unused thread
	{
		EO[0] = t8[2][0] * src[2 * line] + t8[6][0] * src[6 * line];
		EO[1] = t8[2][1] * src[2 * line] + t8[6][1] * src[6 * line];
		EE[0] = t8[0][0] * src[0] + t8[4][0] * src[4 * line];
		EE[1] = t8[0][1] * src[0] + t8[4][1] * src[4 * line];

		/* Combining even and odd terms at each hierarchy levels to calculate the final spatial domain vector */
		E[0] = EE[0] + EO[0];
		E[3] = EE[0] - EO[0];
		E[1] = EE[1] + EO[1];
		E[2] = EE[1] - EO[1];
	}
	__syncthreads();
	if (k < 4)
	{
		dst[k] = (int16_t)g_x265_clip3(-32768, 32767, (E[k] + O[k] + add) >> shift);
		dst[k + 4] = (int16_t)g_x265_clip3(-32768, 32767, (E[3 - k] - O[3 - k] + add) >> shift);
	}
}

__global__ void partialButterflyInverse16(const int16_t* src, int16_t* dst, int shift, int line)
{
	int j = blockIdx.x;// max 16
	int k = threadIdx.x;//max 16
	__shared__ int E[8], O[8];
	__shared__ int EE[4], EO[4];
	__shared__ int EEE[2], EEO[2];
	int add = 1 << (shift - 1);

	dst += 16 * j;
	src += j;

	/* Utilizing symmetry properties to the maximum to minimize the number of multiplications */
	if (k < 8)
	{
		O[k] = t16[1][k] * src[line] + t16[3][k] * src[3 * line] + t16[5][k] * src[5 * line] + t16[7][k] * src[7 * line] +
			t16[9][k] * src[9 * line] + t16[11][k] * src[11 * line] + t16[13][k] * src[13 * line] + t16[15][k] * src[15 * line];
	}
	else if (k < 12)
	{
		int i = k - 8;
		EO[i] = t16[2][i] * src[2 * line] + t16[6][i] * src[6 * line] + t16[10][i]
			* src[10 * line] + t16[14][i] * src[14 * line];
	}
	else if (k == 12){
		EEO[0] = t16[4][0] * src[4 * line] + t16[12][0] * src[12 * line];
		EEE[0] = t16[0][0] * src[0] + t16[8][0] * src[8 * line];
		EEO[1] = t16[4][1] * src[4 * line] + t16[12][1] * src[12 * line];
		EEE[1] = t16[0][1] * src[0] + t16[8][1] * src[8 * line];
	}

	__syncthreads();
	/* Combining even and odd terms at each hierarchy levels to calculate the final spatial domain vector */
	if (k < 2)
	{
		EE[k] = EEE[k] + EEO[k];
		EE[k + 2] = EEE[1 - k] - EEO[1 - k];
	}
	__syncthreads();
	if (k < 4)
	{
		E[k] = EE[k] + EO[k];
		E[k + 4] = EE[3 - k] - EO[3 - k];
	}
	__syncthreads();
	if (k < 8)
	{
		dst[k] = (int16_t)g_x265_clip3(-32768, 32767, (E[k] + O[k] + add) >> shift);
		dst[k + 8] = (int16_t)g_x265_clip3(-32768, 32767, (E[7 - k] - O[7 - k] + add) >> shift);
	}
}

__global__ void partialButterflyInverse32(const int16_t* src, int16_t* dst, int shift, int line)
{
	int j = blockIdx.x;// max 32
	int k = threadIdx.x;//max 32
	__shared__ int E[16], O[16];
	__shared__ int EE[8], EO[8];
	__shared__ int EEE[4], EEO[4];
	__shared__ int EEEE[2], EEEO[2];
	int add = 1 << (shift - 1);

	dst += 32 * j;
	src += j;

	/* Utilizing symmetry properties to the maximum to minimize the number of multiplications */
	if (k < 16)
	{
		O[k] = t32[1][k] * src[line] + t32[3][k] * src[3 * line] + t32[5][k] * src[5 * line] + t32[7][k] * src[7 * line] +
			t32[9][k] * src[9 * line] + t32[11][k] * src[11 * line] + t32[13][k] * src[13 * line] + t32[15][k] * src[15 * line] +
			t32[17][k] * src[17 * line] + t32[19][k] * src[19 * line] + t32[21][k] * src[21 * line] + t32[23][k] * src[23 * line] +
			t32[25][k] * src[25 * line] + t32[27][k] * src[27 * line] + t32[29][k] * src[29 * line] + t32[31][k] * src[31 * line];
	}
	else if (k < 24)
	{
		int i = k - 16;
		EO[i] = t32[2][i] * src[2 * line] + t32[6][i] * src[6 * line] + t32[10][i] * src[10 * line] + t32[14][i] * src[14 * line] +
			t32[18][i] * src[18 * line] + t32[22][i] * src[22 * line] + t32[26][i] * src[26 * line] + t32[30][i] * src[30 * line];
	}

	else if (k < 28)
	{
		int i = k - 24;
		EEO[i] = t32[4][i] * src[4 * line] + t32[12][i] * src[12 * line] + t32[20][i] * src[20 * line] + t32[28][i] * src[28 * line];
	}
	else if (k == 28)
	{
		EEEO[0] = t32[8][0] * src[8 * line] + t32[24][0] * src[24 * line];
		EEEO[1] = t32[8][1] * src[8 * line] + t32[24][1] * src[24 * line];
		EEEE[0] = t32[0][0] * src[0] + t32[16][0] * src[16 * line];
		EEEE[1] = t32[0][1] * src[0] + t32[16][1] * src[16 * line];

		/* Combining even and odd terms at each hierarchy levels to calculate the final spatial domain vector */
		EEE[0] = EEEE[0] + EEEO[0];
		EEE[3] = EEEE[0] - EEEO[0];
		EEE[1] = EEEE[1] + EEEO[1];
		EEE[2] = EEEE[1] - EEEO[1];
	}
	__syncthreads();
	if (k < 4)
	{
		EE[k] = EEE[k] + EEO[k];
		EE[k + 4] = EEE[3 - k] - EEO[3 - k];
	}
	__syncthreads();
	if (k < 8)
	{
		E[k] = EE[k] + EO[k];
		E[k + 8] = EE[7 - k] - EO[7 - k];
	}
	__syncthreads();
	if (k < 16)
	{
		dst[k] = (int16_t)g_x265_clip3(-32768, 32767, (E[k] + O[k] + add) >> shift);
		dst[k + 16] = (int16_t)g_x265_clip3(-32768, 32767, (E[15 - k] - O[15 - k] + add) >> shift);
	}
}

__global__ void partialButterfly4(const int16_t* src, int16_t* dst, int shift, int line)
{
	int j = threadIdx.x;
	int E[2], O[2];
	int add = 1 << (shift - 1);

	src += 4 * j;
	dst += j;

	/* E and O */
	E[0] = src[0] + src[3];
	O[0] = src[0] - src[3];
	E[1] = src[1] + src[2];
	O[1] = src[1] - src[2];

	dst[0] = (int16_t)((t4[0][0] * E[0] + t4[0][1] * E[1] + add) >> shift);
	dst[2 * line] = (int16_t)((t4[2][0] * E[0] + t4[2][1] * E[1] + add) >> shift);
	dst[line] = (int16_t)((t4[1][0] * O[0] + t4[1][1] * O[1] + add) >> shift);
	dst[3 * line] = (int16_t)((t4[3][0] * O[0] + t4[3][1] * O[1] + add) >> shift);
}

//template<FTYPESTAGE type>
//__global__ void transformAtomic(const int16_t *src, int16_t *dst, int n, const int shift) {
//	int col = blockIdx.x;
//	int row = threadIdx.x;
//	int mid = threadIdx.y;
//	extern __shared__ int16_t shared[];
//	shared[row * n + mid] = src[row * n + mid];
//	int size = n*n;
//	int size2 = size * 2;
//	int size3 = size * 3;
//	//sum
//	int *sum = (int*)&shared[size2 + row];
//	sum[row] = 0;
//	//counter
//	int *count = (int*)&shared[size3 + row];
//	count[row] = 0;
//	switch (type)
//	{
//	case DSTN:
//	case DSTT:
//	case IDSTT:
//	case IDSTN:
//		shared[size + row*n + mid] = ta[row][mid];
//		break;
//	case DCT4N:
//	case DCT4T:
//	case IDCT4N:
//	case IDCT4T:
//		shared[size + row*n + mid] = t4[row][mid];
//		break;
//	case DCT8N:
//	case DCT8T:
//	case IDCT8N:
//	case IDCT8T:
//		shared[size + row*n + mid] = t8[row][mid];
//		break;
//	case DCT16N:
//	case DCT16T:
//	case IDCT16N:
//	case IDCT16T:
//		shared[size + row*n + mid] = t16[row][mid];
//		break;
//	case DCT32N:
//	case DCT32T:
//	case IDCT32N:
//	case IDCT32T:
//		shared[size + row*n + mid] = t32[row][mid];
//		break;
//	}
//	__syncthreads();
//
//	int mul;
//	switch (type)
//	{
//	case DSTN:
//	case DCT4N:
//	case DCT8N:
//	case DCT16N:
//	case DCT32N:
//		mul = shared[size + row*n + mid] * shared[mid * n + col];
//		break;
//	case DSTT:
//	case DCT4T:
//	case DCT8T:
//	case DCT16T:
//	case DCT32T:
//		mul = shared[row * n + mid] * shared[size + col*n + mid];
//		break;
//	case IDSTN:
//	case IDCT4N:
//	case IDCT8N:
//	case IDCT16N:
//	case IDCT32N:
//		mul = shared[size + mid*n + row] * shared[mid * n + col];
//		break;
//	case IDSTT:
//	case IDCT4T:
//	case IDCT8T:
//	case IDCT16T:
//	case IDCT32T:
//		mul = shared[row * n + mid] * shared[size + mid*n + col];
//		break;
//	default:
//		mul = 0;
//	}
//	//sum
//	atomicAdd(&sum[row], mul);
//	//counter
//	atomicAdd(&count[row], 1);
//	if (count[row] >= n){
//		sum[row] >>= shift;
//		dst[row * n + col] = (int16_t)sum[row];
//	}
//}

template<FTYPE ftype>
__global__ void transform1stepBatch(const int16_t *src, int16_t *dst, const int n, const int shift1, const int shift2) {
	int row = threadIdx.y;
	int col = threadIdx.x;
	int m = blockIdx.x;
	int size = n*n;
	extern __shared__ int16_t shared[];
	shared[row * n + col] = src[m*size+row * n + col];
	int size2 = size * 2;
	switch (ftype)
	{
	case DST:
	case IDST:
		shared[size + row*n + col] = ta[row][col];
		break;
	case DCT4:
	case IDCT4:
		shared[size + row*n + col] = t4[row][col];
		break;
	case DCT8:
	case IDCT8:
		shared[size + row*n + col] = t8[row][col];
		break;
	case DCT16:
	case IDCT16:
		shared[size + row*n + col] = t16[row][col];
		break;
	case DCT32:
	case IDCT32:
		shared[size + row*n + col] = t32[row][col];
		break;
	}
	__syncthreads();
	int sum = 0;
	for (int i = 0; i < n; i++)
	{
		switch (ftype)
		{
		case DST:
		case DCT4:
		case DCT8:
		case DCT16:
		case DCT32:
			sum += shared[size + row*n + i] * shared[i * n + col];
			break;
		case IDST:
		case IDCT4:
		case IDCT8:
		case IDCT16:
		case IDCT32:
			sum += shared[size + i*n + row] * shared[i * n + col];
			break;
		}
	}
	sum >>= shift1;
	shared[size2 + row * n + col] = sum;
	__syncthreads();

	for (int i = 0; i < n; i++)
	{
		switch (ftype)
		{
		case DST:
		case DCT4:
		case DCT8:
		case DCT16:
		case DCT32:
			sum += shared[size2 + row * n + i] * shared[size + col*n + i];
			break;
		case IDST:
		case IDCT4:
		case IDCT8:
		case IDCT16:
		case IDCT32:
			sum += shared[size2 + row * n + i] * shared[size + i*n + col];
			break;
		}
	}
	sum >>= shift2;
	dst[m*size + row * n + col] = sum;
}

template<FTYPE ftype>
__global__ void transform1step(const int16_t *src, int16_t *dst, const int n, const int shift1, const int shift2) {
	int row = threadIdx.y;
	int col = threadIdx.x;
	extern __shared__ int16_t shared[];
	shared[row * n + col] = src[row * n + col];
	int size = n*n;
	int size2 = size * 2;
	switch (ftype)
	{
	case DST:
	case IDST:
		shared[size + row*n + col] = ta[row][col];
		break;
	case DCT4:
	case IDCT4:
		shared[size + row*n + col] = t4[row][col];
		break;
	case DCT8:
	case IDCT8:
		shared[size + row*n + col] = t8[row][col];
		break;
	case DCT16:
	case IDCT16:
		shared[size + row*n + col] = t16[row][col];
		break;
	case DCT32:
	case IDCT32:
		shared[size + row*n + col] = t32[row][col];
		break;
	}
	__syncthreads();
	int sum = 0;
	for (int i = 0; i < n; i++)
	{
		switch (ftype)
		{
		case DST:
		case DCT4:
		case DCT8:
		case DCT16:
		case DCT32:
			sum += shared[size + row*n + i] * shared[i * n + col];
			break;
		case IDST:
		case IDCT4:
		case IDCT8:
		case IDCT16:
		case IDCT32:
			sum += shared[size + i*n + row] * shared[i * n + col];
			break;
		}
	}
	sum >>= shift1;
	shared[size2 + row * n + col] = sum;
	__syncthreads();

	for (int i = 0; i < n; i++)
	{
		switch (ftype)
		{
		case DST:
		case DCT4:
		case DCT8:
		case DCT16:
		case DCT32:
			sum += shared[size2 + row * n + i] * shared[size + col*n + i];
			break;
		case IDST:
		case IDCT4:
		case IDCT8:
		case IDCT16:
		case IDCT32:
			sum += shared[size2 + row * n + i] * shared[size + i*n + col];
			break;
		}
	}
	sum >>= shift2;
	dst[row * n + col] = sum;
}

__global__ void dct32_g(const int16_t *src, int16_t *dst ,const int shift1, const int shift2) {
	int row = threadIdx.y;
	int col = threadIdx.x;
	const int n = 32;
	extern __shared__ int16_t shared[];
	shared[row * n + col] = src[row * n + col];
	int size = n*n;
	int size2 = size * 2;
	shared[size + row*n + col] = t32[row][col];
	__syncthreads();
	int sum = 0;
	for (int i = 0; i < n; i++)
	{
		sum += shared[size + row*n + i] * shared[i * n + col];
	}
	sum >>= shift1;
	shared[size2 + row * n + col] = sum;
	__syncthreads();

	for (int i = 0; i < n; i++)
	{
		sum += shared[size2 + row * n + i] * shared[size + col*n + i];
	}
	sum >>= shift2;
	dst[row * n + col] = sum;
}

__global__ void idct32_g(const int16_t *src, int16_t *dst, const int shift1, const int shift2) {
	int row = threadIdx.y;
	int col = threadIdx.x;
	const int n = 32;
	extern __shared__ int16_t shared[];
	shared[row * n + col] = src[row * n + col];
	int size = n*n;
	int size2 = size * 2;
	shared[size + row*n + col] = t32[row][col];
	__syncthreads();
	int sum = 0;
	for (int i = 0; i < n; i++)
	{
		sum += shared[size + i*n + row] * shared[i * n + col];
	}
	sum >>= shift1;
	shared[size2 + row * n + col] = sum;
	__syncthreads();

	for (int i = 0; i < n; i++)
	{
		sum += shared[size2 + row * n + i] * shared[size + i*n + col];
	}
	sum >>= shift2;
	dst[row * n + col] = sum;
}

template<FTYPESTAGE type>
__global__ void transform(const int16_t *src, int16_t *dst, int n, const int shift) {
	int row = threadIdx.y;
	int col = threadIdx.x;
	extern __shared__ int16_t shared[];
	shared[row * n + col] = src[row * n + col];
	int size = n*n;
	switch (type)
	{
	case DSTN:
	case DSTT:
	case IDSTT:
	case IDSTN:
		shared[size + row*n + col] = ta[row][col];
		break;
	case DCT4N:
	case DCT4T:
	case IDCT4N:
	case IDCT4T:
		shared[size + row*n + col] = t4[row][col];
		break;
	case DCT8N:
	case DCT8T:
	case IDCT8N:
	case IDCT8T:
		shared[size + row*n + col] = t8[row][col];
		break;
	case DCT16N:
	case DCT16T:
	case IDCT16N:
	case IDCT16T:
		shared[size + row*n + col] = t16[row][col];
		break;
	case DCT32N:
	case DCT32T:
	case IDCT32N:
	case IDCT32T:
		shared[size + row*n + col] = t32[row][col];
		break;
	}
	__syncthreads();
	int sum = 0;
	for (int i = 0; i < n; i++)
	{
		switch (type)
		{
		case DSTN:
		case DCT4N:
		case DCT8N:
		case DCT16N:
		case DCT32N:
			sum += shared[size + row*n + i] * shared[i * n + col];
			break;
		case DSTT:
		case DCT4T:
		case DCT8T:
		case DCT16T:
		case DCT32T:
			sum += shared[row * n + i] * shared[size + col*n + i];
			break;
		case IDSTN:
		case IDCT4N:
		case IDCT8N:
		case IDCT16N:
		case IDCT32N:
			sum += shared[size + i*n + row] * shared[i * n + col];
			break;
		case IDSTT:
		case IDCT4T:
		case IDCT8T:
		case IDCT16T:
		case IDCT32T:
			sum += shared[row * n + i] * shared[size + i*n + col];
			break;
		}
	}
	sum >>= shift;
	dst[row * n + col] = sum;
}

template<FTYPESTAGE type>
__global__ void transformPlain(const int16_t *src, int16_t *dst, int n, const int shift) {
	int row = threadIdx.y;
	int col = threadIdx.x;
	int sum = 0;
	for (int i = 0; i < n; i++)
	{
		switch (type)
		{
		case DSTN:
			sum += ta[row][i] * src[i * n + col];
			break;
		case DSTT:
			sum += src[row * n + i] * ta[col][i];
			break;
		case DCT4N:
			sum += t4[row][i] * src[i * n + col];
			break;
		case DCT4T:
			sum += src[row * n + i] * t4[col][i];
			break;
		case DCT8N:
			sum += t8[row][i] * src[i * n + col];
			break;
		case DCT8T:
			sum += src[row * n + i] * t8[col][i];
			break;
		case DCT16N:
			sum += t16[row][i] * src[i * n + col];
			break;
		case DCT16T:
			sum += src[row * n + i] * t16[col][i];
			break;
		case DCT32N:
			sum += t32[row][i] * src[i * n + col];
			break;
		case DCT32T:
			sum += src[row * n + i] * t32[col][i];
			break;
		case IDSTN:
			sum += ta[i][row] * src[i * n + col];
			break;
		case IDSTT:
			sum += src[row * n + i] * ta[i][col];
			break;
		case IDCT4N:
			sum += t4[i][row] * src[i * n + col];
			break;
		case IDCT4T:
			sum += src[row * n + i] * t4[i][col];
			break;
		case IDCT8N:
			sum += t8[i][row] * src[i * n + col];
			break;
		case IDCT8T:
			sum += src[row * n + i] * t8[i][col];
			break;
		case IDCT16N:
			sum += t16[i][row] * src[i * n + col];
			break;
		case IDCT16T:
			sum += src[row * n + i] * t16[i][col];
			break;
		case IDCT32N:
			sum += t32[i][row] * src[i * n + col];
			break;
		case IDCT32T:
			sum += src[row * n + i] * t32[i][col];
			break;
		}
	}
	sum >>= shift;
	dst[row * n + col] = sum;
}

template __global__ void transformPlain<DSTN>(const int16_t *src, int16_t *dst, int n, const int shift);
template __global__ void transformPlain<DSTT>(const int16_t *src, int16_t *dst, int n, const int shift);
template __global__ void transformPlain<DCT4N>(const int16_t *src, int16_t *dst, int n, const int shift);
template __global__ void transformPlain<DCT4T>(const int16_t *src, int16_t *dst, int n, const int shift);
template __global__ void transformPlain<DCT8N>(const int16_t *src, int16_t *dst, int n, const int shift);
template __global__ void transformPlain<DCT8T>(const int16_t *src, int16_t *dst, int n, const int shift);
template __global__ void transformPlain<DCT16N>(const int16_t *src, int16_t *dst, int n, const int shift);
template __global__ void transformPlain<DCT16T>(const int16_t *src, int16_t *dst, int n, const int shift);
template __global__ void transformPlain<DCT32N>(const int16_t *src, int16_t *dst, int n, const int shift);
template __global__ void transformPlain<DCT32T>(const int16_t *src, int16_t *dst, int n, const int shift);
template __global__ void transformPlain<IDSTN>(const int16_t *src, int16_t *dst, int n, const int shift);
template __global__ void transformPlain<IDSTT>(const int16_t *src, int16_t *dst, int n, const int shift);
template __global__ void transformPlain<IDCT4N>(const int16_t *src, int16_t *dst, int n, const int shift);
template __global__ void transformPlain<IDCT4T>(const int16_t *src, int16_t *dst, int n, const int shift);
template __global__ void transformPlain<IDCT8N>(const int16_t *src, int16_t *dst, int n, const int shift);
template __global__ void transformPlain<IDCT8T>(const int16_t *src, int16_t *dst, int n, const int shift);
template __global__ void transformPlain<IDCT16N>(const int16_t *src, int16_t *dst, int n, const int shift);
template __global__ void transformPlain<IDCT16T>(const int16_t *src, int16_t *dst, int n, const int shift);
template __global__ void transformPlain<IDCT32N>(const int16_t *src, int16_t *dst, int n, const int shift);
template __global__ void transformPlain<IDCT32T>(const int16_t *src, int16_t *dst, int n, const int shift);

template __global__ void transform<DSTN>(const int16_t *src, int16_t *dst, int n, const int shift);
template __global__ void transform<DSTT>(const int16_t *src, int16_t *dst, int n, const int shift);
template __global__ void transform<DCT4N>(const int16_t *src, int16_t *dst, int n, const int shift);
template __global__ void transform<DCT4T>(const int16_t *src, int16_t *dst, int n, const int shift);
template __global__ void transform<DCT8N>(const int16_t *src, int16_t *dst, int n, const int shift);
template __global__ void transform<DCT8T>(const int16_t *src, int16_t *dst, int n, const int shift);
template __global__ void transform<DCT16N>(const int16_t *src, int16_t *dst, int n, const int shift);
template __global__ void transform<DCT16T>(const int16_t *src, int16_t *dst, int n, const int shift);
template __global__ void transform<DCT32N>(const int16_t *src, int16_t *dst, int n, const int shift);
template __global__ void transform<DCT32T>(const int16_t *src, int16_t *dst, int n, const int shift);
template __global__ void transform<IDSTN>(const int16_t *src, int16_t *dst, int n, const int shift);
template __global__ void transform<IDSTT>(const int16_t *src, int16_t *dst, int n, const int shift);
template __global__ void transform<IDCT4N>(const int16_t *src, int16_t *dst, int n, const int shift);
template __global__ void transform<IDCT4T>(const int16_t *src, int16_t *dst, int n, const int shift);
template __global__ void transform<IDCT8N>(const int16_t *src, int16_t *dst, int n, const int shift);
template __global__ void transform<IDCT8T>(const int16_t *src, int16_t *dst, int n, const int shift);
template __global__ void transform<IDCT16N>(const int16_t *src, int16_t *dst, int n, const int shift);
template __global__ void transform<IDCT16T>(const int16_t *src, int16_t *dst, int n, const int shift);
template __global__ void transform<IDCT32N>(const int16_t *src, int16_t *dst, int n, const int shift);
template __global__ void transform<IDCT32T>(const int16_t *src, int16_t *dst, int n, const int shift);

template __global__ void transform1step<DST>(const int16_t *src, int16_t *dst, int n, const int shift1, const int shift2);
template __global__ void transform1step<DCT4>(const int16_t *src, int16_t *dst, int n, const int shift1, const int shift2);
template __global__ void transform1step<DCT8>(const int16_t *src, int16_t *dst, int n, const int shift1, const int shift2);
template __global__ void transform1step<DCT16>(const int16_t *src, int16_t *dst, int n, const int shift1, const int shift2);
template __global__ void transform1step<DCT32>(const int16_t *src, int16_t *dst, int n, const int shift1, const int shift2);
template __global__ void transform1step<IDST>(const int16_t *src, int16_t *dst, int n, const int shift1, const int shift2);
template __global__ void transform1step<IDCT4>(const int16_t *src, int16_t *dst, int n, const int shift1, const int shift2);
template __global__ void transform1step<IDCT8>(const int16_t *src, int16_t *dst, int n, const int shift1, const int shift2);
template __global__ void transform1step<IDCT16>(const int16_t *src, int16_t *dst, int n, const int shift1, const int shift2);
template __global__ void transform1step<IDCT32>(const int16_t *src, int16_t *dst, int n, const int shift1, const int shift2);

template __global__ void transform1stepBatch<DST>(const int16_t *src, int16_t *dst, int n, const int shift1, const int shift2);
template __global__ void transform1stepBatch<DCT4>(const int16_t *src, int16_t *dst, int n, const int shift1, const int shift2);
template __global__ void transform1stepBatch<DCT8>(const int16_t *src, int16_t *dst, int n, const int shift1, const int shift2);
template __global__ void transform1stepBatch<DCT16>(const int16_t *src, int16_t *dst, int n, const int shift1, const int shift2);
template __global__ void transform1stepBatch<DCT32>(const int16_t *src, int16_t *dst, int n, const int shift1, const int shift2);
template __global__ void transform1stepBatch<IDST>(const int16_t *src, int16_t *dst, int n, const int shift1, const int shift2);
template __global__ void transform1stepBatch<IDCT4>(const int16_t *src, int16_t *dst, int n, const int shift1, const int shift2);
template __global__ void transform1stepBatch<IDCT8>(const int16_t *src, int16_t *dst, int n, const int shift1, const int shift2);
template __global__ void transform1stepBatch<IDCT16>(const int16_t *src, int16_t *dst, int n, const int shift1, const int shift2);
template __global__ void transform1stepBatch<IDCT32>(const int16_t *src, int16_t *dst, int n, const int shift1, const int shift2);

template<FTYPE type>
void gpuTransformPlain(const int16_t* h_src, int16_t* h_dst, int shift1, int shift2, int n) {
	// Allocate 3 arrays on GPU
	//int16_t *dst, *src, *src2;
	//hipMalloc(&src, n * n * sizeof(int16_t));
	//hipMalloc(&src2, n * n * sizeof(int16_t));
	//hipMalloc(&dst, n * n * sizeof(int16_t));

	hipMemcpy(src, h_src, n * n * sizeof(int16_t), hipMemcpyHostToDevice);

	dim3 dimGrid(1, 1);
	dim3  dimBlock(n, n);
	switch (type)
	{
	case DST:
		transformPlain<DSTN> << <dimGrid, dimBlock >> >(src, src2, n, shift1);
		hipDeviceSynchronize();
		transformPlain<DSTT> << <dimGrid, dimBlock >> >(src2, dst, n, shift2);
		break;
	case DCT4:
		transformPlain<DCT4N> << <dimGrid, dimBlock >> >(src, src2, n, shift1);
		hipDeviceSynchronize();
		transformPlain<DCT4T> << <dimGrid, dimBlock >> >(src2, dst, n, shift2);
		break;
	case DCT8:
		transformPlain<DCT8N> << <dimGrid, dimBlock >> >(src, src2, n, shift1);
		hipDeviceSynchronize();
		transformPlain<DCT8T> << <dimGrid, dimBlock >> >(src2, dst, n, shift2);
		break;
	case DCT16:
		transformPlain<DCT16N> << <dimGrid, dimBlock>> >(src, src2, n, shift1);
		hipDeviceSynchronize();
		transformPlain<DCT16T> << <dimGrid, dimBlock>> >(src2, dst, n, shift2);
		break;
	case DCT32:
		transformPlain<DCT32N> << <dimGrid, dimBlock >> >(src, src2, n, shift1);
		hipDeviceSynchronize();
		transformPlain<DCT32T> << <dimGrid, dimBlock >> >(src2, dst, n, shift2);
		break;
	case IDST:
		transformPlain<IDSTN> << <dimGrid, dimBlock >> >(src, src2, n, shift1);
		hipDeviceSynchronize();
		transformPlain<IDSTT> << <dimGrid, dimBlock >> >(src2, dst, n, shift2);
		break;
	case IDCT4:
		transformPlain<IDCT4N> << <dimGrid, dimBlock >> >(src, src2, n, shift1);
		hipDeviceSynchronize();
		transformPlain<IDCT4T> << <dimGrid, dimBlock >> >(src2, dst, n, shift2);
		break;
	case IDCT8:
		transformPlain<IDCT8N> << <dimGrid, dimBlock >> >(src, src2, n, shift1);
		hipDeviceSynchronize();
		transformPlain<IDCT8T> << <dimGrid, dimBlock >> >(src2, dst, n, shift2);
		break;
	case IDCT16:
		transformPlain<IDCT16N> << <dimGrid, dimBlock >> >(src, src2, n, shift1);
		hipDeviceSynchronize();
		transformPlain<IDCT16T> << <dimGrid, dimBlock >> >(src2, dst, n, shift2);
		break;
	case IDCT32:
		transformPlain<IDCT32N> << <dimGrid, dimBlock >> >(src, src2, n, shift1);
		hipDeviceSynchronize();
		transformPlain<IDCT32T> << <dimGrid, dimBlock >> >(src2, dst, n, shift2);
		break;
	}

	// Copy (and print) the result on host memory
	hipMemcpy(h_dst, dst, n * n * sizeof(int16_t), hipMemcpyDeviceToHost);

	//Free GPU memory
	//hipFree(src);
	//hipFree(src2);
	//hipFree(dst);
}

template<FTYPE type>
void gpuTransformShared(const int16_t* h_src, int16_t* h_dst, int shift1, int shift2, int n) {
	// Allocate 3 arrays on GPU
	//int16_t *dst, *src, *src2;
	//hipMalloc(&src, n * n * sizeof(int16_t));
	//hipMalloc(&src2, n * n * sizeof(int16_t));
	//hipMalloc(&dst, n * n * sizeof(int16_t));

	hipMemcpy(src, h_src, n * n * sizeof(int16_t), hipMemcpyHostToDevice);

	dim3 dimGrid(1);
	dim3  dimBlock(n, n);
	int size = 2 * n*n* sizeof(int16_t);
	switch (type)
	{
	case DST:
		transform<DSTN> << <dimGrid, dimBlock, size >> >(src, src2, n, shift1);
		hipDeviceSynchronize();
		transform<DSTT> << <dimGrid, dimBlock, size >> >(src2, dst, n, shift2);
		break;
	case DCT4:
		transform<DCT4N> << <dimGrid, dimBlock, size >> >(src, src2, n, shift1);
		hipDeviceSynchronize();
		transform<DCT4T> << <dimGrid, dimBlock, size >> >(src2, dst, n, shift2);
		break;
	case DCT8:
		transform<DCT8N> << <dimGrid, dimBlock, size >> >(src, src2, n, shift1);
		hipDeviceSynchronize();
		transform<DCT8T> << <dimGrid, dimBlock, size >> >(src2, dst, n, shift2);
		break;
	case DCT16:
		transform<DCT16N> << <dimGrid, dimBlock, size >> >(src, src2, n, shift1);
		hipDeviceSynchronize();
		transform<DCT16T> << <dimGrid, dimBlock, size >> >(src2, dst, n, shift2);
		break;
	case DCT32:
		transform<DCT32N> << <dimGrid, dimBlock, size >> >(src, src2, n, shift1);
		hipDeviceSynchronize();
		transform<DCT32T> << <dimGrid, dimBlock, size >> >(src2, dst, n, shift2);
		break;
	case IDST:
		transform<IDSTN> << <dimGrid, dimBlock, size >> >(src, src2, n, shift1);
		hipDeviceSynchronize();
		transform<IDSTT> << <dimGrid, dimBlock, size >> >(src2, dst, n, shift2);
		break;
	case IDCT4:
		transform<IDCT4N> << <dimGrid, dimBlock, size >> >(src, src2, n, shift1);
		hipDeviceSynchronize();
		transform<IDCT4T> << <dimGrid, dimBlock, size >> >(src2, dst, n, shift2);
		break;
	case IDCT8:
		transform<IDCT8N> << <dimGrid, dimBlock, size >> >(src, src2, n, shift1);
		hipDeviceSynchronize();
		transform<IDCT8T> << <dimGrid, dimBlock, size >> >(src2, dst, n, shift2);
		break;
	case IDCT16:
		transform<IDCT16N> << <dimGrid, dimBlock, size >> >(src, src2, n, shift1);
		hipDeviceSynchronize();
		transform<IDCT16T> << <dimGrid, dimBlock, size >> >(src2, dst, n, shift2);
		break;
	case IDCT32:
		transform<IDCT32N> << <dimGrid, dimBlock, size >> >(src, src2, n, shift1);
		hipDeviceSynchronize();
		transform<IDCT32T> << <dimGrid, dimBlock, size >> >(src2, dst, n, shift2);
		break;
	}

	// Copy (and print) the result on host memory
	hipMemcpy(h_dst, dst, n * n * sizeof(int16_t), hipMemcpyDeviceToHost);

	//Free GPU memory
	//hipFree(src);
	//hipFree(src2);
	//hipFree(dst);
}

template<FTYPE type>
void gpuTransform1Step(const int16_t* h_src, int16_t* h_dst, int shift1, int shift2, int n) {
	// Allocate 3 arrays on GPU
	//int16_t *dst, *src;
	//hipMalloc(&src, n * n * sizeof(int16_t));
	//hipMalloc(&dst, n * n * sizeof(int16_t));

	hipMemcpy(src, h_src, n * n * sizeof(int16_t), hipMemcpyHostToDevice);

	dim3 dimGrid(1);
	dim3  dimBlock(n, n);
	int size = 3 * n*n* sizeof(int16_t);
	switch (type)
	{
	case DST:
		transform1step<DST> << <dimGrid, dimBlock, size >> >(src, dst, n, shift1, shift2);
		break;
	case DCT4:
		transform1step<DCT4> << <dimGrid, dimBlock, size >> >(src, dst, n, shift1, shift2);
		break;
	case DCT8:
		transform1step<DCT8> << <dimGrid, dimBlock, size >> >(src, dst, n, shift1, shift2);
		break;
	case DCT16:
		transform1step<DCT16> << <dimGrid, dimBlock, size >> >(src, dst, n, shift1, shift2);
		break;
	case DCT32:
		dct32_g << <dimGrid, dimBlock, size >> >(src, dst, shift1, shift2);
		break;
	case IDST:
		transform1step<IDST> << <dimGrid, dimBlock, size >> >(src, dst, n, shift1, shift2);
		break;
	case IDCT4:
		transform1step<IDCT4> << <dimGrid, dimBlock, size >> >(src, dst, n, shift1, shift2);
		break;
	case IDCT8:
		transform1step<IDCT8> << <dimGrid, dimBlock, size >> >(src, dst, n, shift1, shift2);
		break;
	case IDCT16:
		transform1step<IDCT16> << <dimGrid, dimBlock, size >> >(src, dst, n, shift1, shift2);
		break;
	case IDCT32:
		idct32_g<< <dimGrid, dimBlock, size >> >(src, dst, shift1, shift2);
		break;
	}

	// Copy (and print) the result on host memory
	hipMemcpy(h_dst, dst, n * n * sizeof(int16_t), hipMemcpyDeviceToHost);

	//Free GPU memory
	//hipFree(src);
	//hipFree(dst);
}

template<FTYPE ftype>
void gpuTransform1StepBatch(const int16_t* h_src, int16_t* h_dst, int shift1, int shift2, int n,int m) {
	// Allocate 3 arrays on GPU
	//int16_t *dst, *src, *src2;
	//hipMalloc(&src, n * n * sizeof(int16_t));
	//hipMalloc(&src2, n * n * sizeof(int16_t));
	//hipMalloc(&dst, n * n * sizeof(int16_t));

	hipMemcpy(src, h_src, m * n * n * sizeof(int16_t), hipMemcpyHostToDevice);

	dim3 dimGrid(m);
	dim3  dimBlock(n, n);
	int size = 3 * n*n* sizeof(int16_t);
	switch (ftype)
	{
	case DST:
		transform1stepBatch<DST> << <dimGrid, dimBlock, size >> >(src, dst, n, shift1, shift2);
		break;
	case DCT4:
		transform1stepBatch<DCT4> << <dimGrid, dimBlock, size >> >(src, dst, n, shift1, shift2);
		break;
	case DCT8:
		transform1stepBatch<DCT8> << <dimGrid, dimBlock, size >> >(src, dst, n, shift1, shift2);
		break;
	case DCT16:
		transform1stepBatch<DCT16> << <dimGrid, dimBlock, size >> >(src, dst, n, shift1, shift2);
		break;
	case DCT32:
		transform1stepBatch<DCT32> << <dimGrid, dimBlock, size >> >(src, dst, n, shift1, shift2);
		break;
	case IDST:
		transform1stepBatch<IDST> << <dimGrid, dimBlock, size >> >(src, dst, n, shift1, shift2);
		break;
	case IDCT4:
		transform1stepBatch<IDCT4> << <dimGrid, dimBlock, size >> >(src, dst, n, shift1, shift2);
		break;
	case IDCT8:
		transform1stepBatch<IDCT8> << <dimGrid, dimBlock, size >> >(src, dst, n, shift1, shift2);
		break;
	case IDCT16:
		transform1stepBatch<IDCT16> << <dimGrid, dimBlock, size >> >(src, dst, n, shift1, shift2);
		break;
	case IDCT32:
		transform1stepBatch<IDCT32> << <dimGrid, dimBlock, size >> >(src, dst,n, shift1, shift2);
		break;
	}

	// Copy (and print) the result on host memory
	hipMemcpy(h_dst, dst, m * n * n * sizeof(int16_t), hipMemcpyDeviceToHost);

	//Free GPU memory
	//hipFree(src);
	//hipFree(src2);
	//hipFree(dst);
}

template<FTYPE type>
void gpuLessMulTransform(const int16_t* h_src, int16_t* h_dst, int shift1, int shift2, int n) {
	// Allocate 3 arrays on GPU
	//int16_t *dst, *src, *src2;
	//hipMalloc(&src, n * n * sizeof(int16_t));
	//hipMalloc(&src2, n * n * sizeof(int16_t));
	//hipMalloc(&dst, n * n * sizeof(int16_t));

	hipMemcpy(src, h_src, n * n * sizeof(int16_t), hipMemcpyHostToDevice);

	dim3 dimGrid(n);
	dim3  dimBlock(n);

	switch (type)
	{
	case DST:
		fastForwardDst << <1, n>> >(src, src2,shift1);
		hipDeviceSynchronize();
		fastForwardDst << <1, n >> >(src2, dst, shift2);
		break;
	case DCT4:
		partialButterfly4 << <1, n >> >(src, src2, shift1, n);
		hipDeviceSynchronize();
		partialButterfly4 << <1, n >> >(src2, dst, shift2, n);
		break;
	case DCT8:
		partialButterfly8 << <dimGrid, dimBlock >> >(src, src2, shift1, n);
		hipDeviceSynchronize();
		partialButterfly8 << <dimGrid, dimBlock >> >(src2, dst, shift2, n);
		break;
	case DCT16:
		partialButterfly16 << <dimGrid, dimBlock >> >(src, src2, shift1, n);
		hipDeviceSynchronize();
		partialButterfly16 << <dimGrid, dimBlock >> >(src2, dst, shift2, n);
		break;
	case DCT32:
		partialButterfly32 << <dimGrid, dimBlock >> >(src, src2, shift1, n);
		hipDeviceSynchronize();
		partialButterfly32 << <dimGrid, dimBlock >> >(src2, dst, shift2, n);
		break;
	case IDST:
		inversedst << <1, n >> >(src, src2, shift1);
		hipDeviceSynchronize();
		inversedst << <1, n >> >(src2, dst, shift2);
		break;
	case IDCT4:
		partialButterflyInverse4 << <1, n >> >(src, src2, shift1, n);
		hipDeviceSynchronize();
		partialButterflyInverse4 << <1, n >> >(src2, dst, shift2, n);
		break;
	case IDCT8:
		partialButterflyInverse8 << <dimGrid, dimBlock >> >(src, src2, shift1, n);
		hipDeviceSynchronize();
		partialButterflyInverse8 << <dimGrid, dimBlock >> >(src2, dst, shift2, n);
		break;
	case IDCT16:
		partialButterflyInverse16 << <dimGrid, dimBlock >> >(src, src2, shift1, n);
		hipDeviceSynchronize();
		partialButterflyInverse16 << <dimGrid, dimBlock >> >(src2, dst, shift2, n);
		break;
	case IDCT32:
		partialButterflyInverse32 << <dimGrid, dimBlock >> >(src, src2, shift1, n);
		hipDeviceSynchronize();
		partialButterflyInverse32 << <dimGrid, dimBlock >> >(src2, dst, shift2, n);
		break;
	}

	// Copy (and print) the result on host memory
	hipMemcpy(h_dst, dst, n * n * sizeof(int16_t), hipMemcpyDeviceToHost);

	//Free GPU memory
	//hipFree(src);
	//hipFree(src2);
	//hipFree(dst);
}

void gpuTransform(const int16_t* h_src, int16_t* h_dst, int shift1, int shift2, int n, FTYPE type) {
	int cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return;
	}
}